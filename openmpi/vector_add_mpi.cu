#include "hip/hip_runtime.h"
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1024

__global__ void vector_add(float *a, float *b, float *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        c[i] = a[i] + b[i];
}

void check_cuda(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s: %s\n", msg, hipGetErrorString(err));
        MPI_Abort(MPI_COMM_WORLD, -1);
    }
}

int main(int argc, char **argv) {
    MPI_Init(&argc, &argv);
    int rank, size;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    // Set device by rank
    check_cuda(hipSetDevice(rank), "hipSetDevice");

    float *a, *b, *c;
    float *d_a, *d_b, *d_c;

    // Allocate host memory
    a = (float*)malloc(N * sizeof(float));
    b = (float*)malloc(N * sizeof(float));
    c = (float*)malloc(N * sizeof(float));

    // Initialize input vectors
    for (int i = 0; i < N; ++i) {
        a[i] = rank + 1.0f;
        b[i] = i * 1.0f;
    }

    // Allocate device memory
    check_cuda(hipMalloc(&d_a, N * sizeof(float)), "hipMalloc d_a");
    check_cuda(hipMalloc(&d_b, N * sizeof(float)), "hipMalloc d_b");
    check_cuda(hipMalloc(&d_c, N * sizeof(float)), "hipMalloc d_c");

    // Copy to device
    check_cuda(hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice), "Memcpy a");
    check_cuda(hipMemcpy(d_b, b, N * sizeof(float), hipMemcpyHostToDevice), "Memcpy b");

    // Launch kernel
    vector_add<<<(N + 255)/256, 256>>>(d_a, d_b, d_c, N);

    // Copy result back
    check_cuda(hipMemcpy(c, d_c, N * sizeof(float), hipMemcpyDeviceToHost), "Memcpy c");

    // Print a few elements from each rank
    printf("Rank %d: c[0]=%f, c[N-1]=%f\n", rank, c[0], c[N-1]);

    // Clean up
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    MPI_Finalize();
    return 0;
}
